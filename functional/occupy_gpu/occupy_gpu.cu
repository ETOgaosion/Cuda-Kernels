#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <hip/hip_runtime.h>
#include <thread>
#include <chrono>
#include <vector>

__global__ void dummyKernel(float *a, int max_iter) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    while (true) {
        a[idx] = sinf(a[idx]);
    }
}

void occupyGPU(int device, float gpuRatio, float memRatio, int max_iter) {
    hipSetDevice(device);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);

    size_t totalMem = prop.totalGlobalMem;
    size_t memToUse = static_cast<size_t>(totalMem * memRatio);

    // 分配显存
    float* d_mem;
    hipMalloc((void**)&d_mem, memToUse);

    // 启动 CUDA 核心函数
    const int blockSize = 64;
    const int SMCount = prop.multiProcessorCount;
    const int total_threads = 2048 * SMCount * gpuRatio;
    int calculatedBlocks = total_threads / blockSize;
    if (calculatedBlocks < 4) {
        calculatedBlocks = 4;
    }
    const int numBlocks = (calculatedBlocks - (calculatedBlocks % 4));
    std::cout << "SM count: " << SMCount << ", numBlocks: " << numBlocks << std::endl;

    while (true) {
        dummyKernel<<<numBlocks, blockSize>>>(d_mem, max_iter);
        hipDeviceSynchronize();
    }

    hipFree(d_mem);
}

int main(int argc, char **argv) {
    float gpuRatio = 0.4, memRatio = 0.4;
    if (argc >= 3) {
        std::cout << "argc: " << argc << ", argv[1]: " << argv[1] << std::endl;
        gpuRatio = (float)std::stoi(argv[1]) / 100.0;
        memRatio = (float)std::stoi(argv[2]) / 100.0;
    }
    std::cout << "GPU ratio: " << gpuRatio << ", memory ratio: " << memRatio << std::endl;

    int deviceIndex = 0;
    if (argc >= 4) {
        deviceIndex = std::stoi(argv[3]);
    }

    int max_iter = 10;
    if (argc >= 5) {
        max_iter = std::stoi(argv[4]);
    }

    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    if (deviceIndex >= deviceCount) {
        std::vector<std::thread> occupyThreads;
        for (int i = 0; i < deviceCount; i++) {
            occupyThreads.emplace_back(occupyGPU(i, gpuRatio, memRatio, max_iter));
        }
        for (auto &it : occupyThreads) {
            it.join();
        }
    }
    else {
        occupyGPU(deviceIndex, gpuRatio, memRatio, max_iter);
    }
    return 0;
}
