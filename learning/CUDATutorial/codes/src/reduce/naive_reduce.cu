#include "hip/hip_runtime.h"
#include <stdio.h>

#include "reduce.cuh"

template <int BLOCK_SIZE>
__global__ void reduce_naive_kernel(float *arr, float *out, int len) {
    __shared__ float sdata[BLOCK_SIZE];
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int bdim = blockDim.x;
    int i = bid * bdim  + tid;
    if (i < len) {
        sdata[tid] = arr[i];
    }
    __syncthreads();
    for (int s = 1; s < bdim; s *= 2) {
        if (tid % (2 * s) == 0 && i + s < len) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        out[bid] = sdata[0];
    }
}

int main() {
    int len = 1024;
    float *arr = new float[len];
    float *out = new float[len];
    randomize_matrix(arr, 1, len);
    float ref_sum = 0.0f;
    for (int i = 0; i < len; i++) {
        ref_sum += arr[i];
    }
    float *d_arr, *d_out;
    hipMalloc(&d_arr, len * sizeof(float));
    hipMalloc(&d_out, len * sizeof(float));
    hipMemcpy(d_arr, arr, len * sizeof(float), hipMemcpyHostToDevice);

    const int block_size = 32;
    const int grid_size = (len + block_size - 1) / block_size;
    reduce_naive_kernel<block_size><<<grid_size, block_size>>>(d_arr, d_out, len);

    hipMemcpy(out, d_out, len * sizeof(float), hipMemcpyDeviceToHost);
    float sum = 0;
    for (int i = 0; i < grid_size; i++) {
        sum += out[i];
    }
    printf("Sum: %f, RefSum: %f\n", sum, ref_sum);
    delete[] arr;
    delete[] out;
    hipFree(d_arr);
    hipFree(d_out);
    return 0;
}